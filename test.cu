#include <stdexcept>
#include <string>
#include "opencv2/imgproc.hpp"
#include "opencv2/highgui.hpp"
#include "opencv2/cudaimgproc.hpp"
#include "opencv2/cudawarping.hpp"
#include "opencv2/core/cuda.hpp"
//#include "opencv2/core.hpp"
//#include "opencv2/core/cuda_types.hpp"
//#include "opencv2/core/cuda.inl.hpp"
#include <sys/time.h>

using namespace std;
using namespace cv;

int main() {

  Mat dst;
  cuda::GpuMat d_src, d_dst, d_xmap, d_ymap;

  int interpolation = INTER_LINEAR;
  int borderMode = BORDER_REPLICATE;

  hipSetDevice(1);

  int gpucount = cuda::getCudaEnabledDeviceCount();
  printf("gpucount: %d\n", gpucount);

  Mat _src = imread("Car-Wallpaper-HD-1080x1920-4.jpg", CV_LOAD_IMAGE_COLOR);
  cuda::HostMem src(_src, cuda::HostMem::PAGE_LOCKED);

  for (int size = 1000; size <= 4000; size *= 2)
  {

    int nrow = src.rows;
    int ncol = src.cols;
    printf("nrow: %d, ncol: %d\n", nrow, ncol);

    cuda::HostMem xmap(size, size, CV_32F, cuda::HostMem::PAGE_LOCKED);
    cuda::HostMem ymap(size, size, CV_32F, cuda::HostMem::PAGE_LOCKED);
    Mat h_xmap = xmap.createMatHeader();
    Mat h_ymap = xmap.createMatHeader();

    for (int i = 0; i < size; ++i)
    {
      float* xmap_row = h_xmap.ptr<float>(i);
      float* ymap_row = h_ymap.ptr<float>(i);
      for (int j = 0; j < size; ++j)
      {
        xmap_row[j] = (j - size * 0.5f) * 0.75f + size * 0.5f;
        ymap_row[j] = (i - size * 0.5f) * 0.75f + size * 0.5f;
      }
    }

    struct timeval start, stop;
    gettimeofday(&start, NULL);
    remap(_src, dst, h_xmap, h_ymap, interpolation, borderMode);
    gettimeofday(&stop, NULL);
    float elapse = stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec) * 1e-6;
    printf("cpu remap time:               %f\n", elapse);

    char fname[256];
    std::sprintf(fname, "output_%d.jpg", size);
    imwrite(fname, dst);

    d_xmap.upload(xmap);
    d_ymap.upload(ymap);

    gettimeofday(&start, NULL);
    d_src.upload(src);
    gettimeofday(&stop, NULL);
    elapse = stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec) * 1e-6;
    printf("gpu upload time:              %f\n", elapse);

    gettimeofday(&start, NULL);
    cuda::remap(d_src, d_dst, d_xmap, d_ymap, interpolation, borderMode);
    gettimeofday(&stop, NULL);
    elapse = stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec) * 1e-6;
    printf("gpu remap  time:              %f\n", elapse);

    cuda::HostMem cudst(size, size, src.type(), cuda::HostMem::PAGE_LOCKED);
    gettimeofday(&start, NULL);
    d_dst.download(cudst.createMatHeader());
    gettimeofday(&stop, NULL);
    elapse = stop.tv_sec - start.tv_sec + (stop.tv_usec - start.tv_usec) * 1e-6;
    printf("gpu download time:            %f\n", elapse);

    std::sprintf(fname, "cuoutput_%d.jpg", size);
    imwrite(fname, cudst.createMatHeader());

  }

  return 0;
}
